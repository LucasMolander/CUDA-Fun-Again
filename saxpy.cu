#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>

#include "util.h"

/**
This is 26.67 billion saxpys per second for N = 200,000,000

It has `n` and `a` as parameters,
and there is a branch on the thread index being < n.
*/

#define N 200'000'000
#define TOTAL_SIZE N * sizeof(float)

#define A 2.3f
#define X_VAL 1.0f
#define Y_VAL 2.0f

#define CUDA_SUCC hipError_t::hipSuccess
#define H_TO_D hipMemcpyKind::hipMemcpyHostToDevice
#define D_TO_H hipMemcpyKind::hipMemcpyDeviceToHost


__global__
void saxpy(uint64_t n, float a, float *d_x, float *d_y, float *d_z) {
  uint64_t tIdx = threadIdx.x + (blockDim.x * blockIdx.x);
  if (tIdx < n) {
    d_z[tIdx] = (a * d_x[tIdx]) + d_y[tIdx];
  }
}


int main(void) {
  printf("SAXPY Version 1\n");

  printf("Total size of a vector: %lld MB\n", TOTAL_SIZE / (1024 * 1024));

  hipDeviceProp_t props;
  if (getDeviceProps(&props) != 0) {
    printf("Unable to get device props\n");
    return 1;
  }

  hipError_t e;
  float *x, *y, *z;       // Host arrays
  float *d_x, *d_y, *d_z; // Device arrays

  // Allocate space on Host
  x = (float*)malloc(TOTAL_SIZE);
  y = (float*)malloc(TOTAL_SIZE);
  z = (float*)malloc(TOTAL_SIZE);

  // Allocate space on Device
  if ((e = hipMalloc(&d_x, TOTAL_SIZE)) != CUDA_SUCC) {
    printf("Failed to allocate device X: %d\n", e);
    return 1;
  }
  if ((e = hipMalloc(&d_y, TOTAL_SIZE)) != CUDA_SUCC) {
    printf("Failed to allocate device Y: %d\n", e);
    return 1;
  }
  if ((e = hipMalloc(&d_z, TOTAL_SIZE)) != CUDA_SUCC) {
    printf("Failed to allocate device Z: %d\n", e);
    return 1;
  }

  // Set the values on Host (Z is for the result)
  for (uint64_t i = 0; i < N; i++) {
    x[i] = X_VAL;
    y[i] = Y_VAL;
  }

  // Copy values for X and Y over to Device (Z is for the result)
  if ((e = hipMemcpy(d_x, x, TOTAL_SIZE, H_TO_D)) != CUDA_SUCC) {
    printf("Failed to copy X to device: %d\n", e);
    return 1;
  }
  if ((e = hipMemcpy(d_y, y, TOTAL_SIZE, H_TO_D)) != CUDA_SUCC) {
    printf("Failed to copy Y to device: %d\n", e);
    return 1;
  }

  // Block size
  int warpSize = props.warpSize;
  uint64_t maxBlockSizeX = props.maxThreadsDim[0];
  int blockSize = warpSize * (maxBlockSizeX / warpSize);

  // Grid size
  uint64_t maxGridSizeX = props.maxGridSize[0];
  uint64_t nBlocks = N / blockSize;
  int nExtraThreads = N % blockSize;
  if (nExtraThreads > 0) {
    ++nBlocks;
  }

  printf("N: %lld\n", (uint64_t)N);
  printf("Warp size: %d\n", warpSize);
  printf("Max block size X: %lld\n", maxBlockSizeX);
  printf("Block size: %d\n", blockSize);
  printf("Max grid size X: %lld\n", maxGridSizeX);
  printf("N blocks: %lld\n", nBlocks);
  printf("N extra threads (last block): %d\n", nExtraThreads);

  if (nBlocks > maxGridSizeX) {
    printf("nBlocks > maxGridSizeX (%lld > %lld)\n", nBlocks, maxGridSizeX);
    return 1;
  }

  printf("A: %f\n", A);

  saxpy<<<nBlocks, blockSize>>>(N, A, d_x, d_y, d_z);

  // Copy Z back to Host
  if ((e = hipMemcpy(z, d_z, TOTAL_SIZE, D_TO_H)) != CUDA_SUCC) {
    printf("Failed to copy Z to host: %d\n", e);
    return 1;
  }

  // Free up the Device memory
  if ((e = hipFree(d_x)) != CUDA_SUCC) {
    printf("Failed to free device X: %d", e);
    return 1;
  }
  if ((e = hipFree(d_y)) != CUDA_SUCC) {
    printf("Failed to free device Y: %d", e);
    return 1;
  }
  if ((e = hipFree(d_z)) != CUDA_SUCC) {
    printf("Failed to free device Z: %d", e);
    return 1;
  }

  // Free up the host memory
  free(x);
  free(y);

  // Do some stuff with the results
  double sum = 0.0;
  for (int i = 0; i < N; i++) {
    sum += z[i];
  }
  double expectedSum = N * ((A * X_VAL) + Y_VAL);
  double error = sum - expectedSum;
  printf("Z sum: %llf\n", sum);
  printf("Expected Z sum: %llf\n", expectedSum);
  printf("Error: %llf\n", error);

  // Now free up the results
  free(z);

  printf("Done!\n");

  return 0;
}
